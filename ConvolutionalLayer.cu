#include "hip/hip_runtime.h"
// This does not really implement a convolution. It just gathers data together
// in prepartion for matrix muliplictation. "Proper convolution" =
// "ConvolutionalLayer" + "NetworkInNetworkLayer"

#include "ConvolutionalLayer.h"
#include <iostream>
#include <vector>
#include <cassert>
#include "utilities.h"
#include "Regions.h"

__global__ void dPropForwardToMatrixMultiplyInput(float *d_features,
                                                  float *d_convolved,
                                                  int *rules, int count,
                                                  int nIn) {
  __shared__ int r[KERNELBLOCKSIZE];
  for (int c = blockIdx.x * KERNELBLOCKSIZE; c < count;
       c += (1 << 12) * KERNELBLOCKSIZE) {
    int n = min(KERNELBLOCKSIZE, count - c);
    r[threadIdx.x] = (threadIdx.x < n) ? rules[c + threadIdx.x] * nIn : 0;
    __syncthreads();
    for (int q = 0; q < n; q++) {
      int rq = r[q];
      int i = (c + q) * nIn;
      for (int j = threadIdx.x; j < nIn; j += KERNELBLOCKSIZE) {
        d_convolved[i + j] = d_features[rq + j];
      }
    }
    __syncthreads();
  }
}
void propForwardToMatrixMultiply(float *inFeatures, float *outFeatures,
                                 int *rules, int count, int nIn,
                                 cudaMemStream &memStream) {
  assert(count > 0);
  int batch = min(1 << 12, (count + KERNELBLOCKSIZE - 1) / KERNELBLOCKSIZE);
  dPropForwardToMatrixMultiplyInput
          << <batch, KERNELBLOCKSIZE, 0, memStream.stream>>>
      (inFeatures, outFeatures, rules, count, nIn);
  cudaCheckError();
}
__global__ void dPropBackwardFromMatrixMultiplyOutput(float *d_deltaGrid,
                                                      float *d_deltaConvolved,
                                                      int *rules, int count,
                                                      int nIn) {
  __shared__ int r[KERNELBLOCKSIZE];
  for (int c = blockIdx.x * KERNELBLOCKSIZE; c < count;
       c += (1 << 12) * KERNELBLOCKSIZE) {
    int n = min(KERNELBLOCKSIZE, count - c);
    r[threadIdx.x] = (threadIdx.x < n) ? rules[c + threadIdx.x] * nIn : 0;
    __syncthreads();
    for (int q = 0; q < n; q++) {
      int rq = r[q];
      int i = (c + q) * nIn;
      for (int j = threadIdx.x; j < nIn; j += KERNELBLOCKSIZE) {
        if (rq >= 0)
          atomicAdd(&d_deltaGrid[rq + j], d_deltaConvolved[i + j]);
      }
    }
    __syncthreads();
  }
}
void propBackwardFromMatrixMultiply(float *inDFeatures, float *outDFeatures,
                                    int *rules, int count, int nIn,
                                    cudaMemStream &memStream) {
  assert(count > 0);
  int batch = min(1 << 12, (count + KERNELBLOCKSIZE - 1) / KERNELBLOCKSIZE);
  dPropBackwardFromMatrixMultiplyOutput
          << <batch, KERNELBLOCKSIZE, 0, memStream.stream>>>
      (inDFeatures, outDFeatures, rules, count, nIn);
  cudaCheckError();
}

template <typename t>
void convolutionFeaturesPresent(std::vector<t> &d_src, std::vector<t> &d_dest,
                                int nf, int nfp, int nCopies) {
  for (int i = 0; i < nfp * nCopies; ++i) {
    d_dest[i] = d_src[i % nfp] + nf * (i / nfp);
  }
}
template void convolutionFeaturesPresent<int>(std::vector<int> &d_src,
                                              std::vector<int> &d_dest, int nf,
                                              int nfp, int nCopies);

ConvolutionalLayer::ConvolutionalLayer(cudaMemStream &memStream, int filterSize,
                                       int filterStride, int dimension,
                                       int nFeaturesIn, int minActiveInputs)
    : SpatiallySparseLayer(memStream), filterSize(filterSize),
      filterStride(filterStride), dimension(dimension),
      nFeaturesIn(nFeaturesIn), minActiveInputs(minActiveInputs) {
  fs = ipow(filterSize, dimension);
  nFeaturesOut = fs * nFeaturesIn;
  std::cout << "Convolution " << filterSize << "^" << dimension << "x"
            << nFeaturesIn << "->" << nFeaturesOut;
  if (filterStride > 1)
    std::cout << " stride:" << filterStride;
  if (minActiveInputs > 1)
    std::cout << " minActiveInputs:" << minActiveInputs;
  std::cout << std::endl;
}
void ConvolutionalLayer::preprocess(SpatiallySparseBatch &batch,
                                    SpatiallySparseBatchInterface &input,
                                    SpatiallySparseBatchInterface &output) {
  output.nFeatures = nFeaturesOut;
  assert(input.nFeatures == nFeaturesIn);
  assert(input.spatialSize >= filterSize);
  assert((input.spatialSize - filterSize) % filterStride == 0);
  output.spatialSize = (input.spatialSize - filterSize) / filterStride + 1;
  output.nSpatialSites = 0;
  output.grids.resize(batch.batchSize);
  output.backpropErrors = input.backpropErrors;
  RegularSquareRegions regions(inSpatialSize, outSpatialSize, dimension,
                               filterSize, filterStride);
  for (int item = 0; item < batch.batchSize; item++) {
    gridRules(input.grids[item], output.grids[item], regions,
              output.nSpatialSites, output.rules.hVector(), true,
              minActiveInputs);
  }
  output.featuresPresent.resize(input.featuresPresent.size() * fs);
  convolutionFeaturesPresent(input.featuresPresent.hVector(),
                             output.featuresPresent.hVector(), input.nFeatures,
                             input.featuresPresent.size(), fs);
}
void ConvolutionalLayer::forwards(SpatiallySparseBatch &batch,
                                  SpatiallySparseBatchInterface &input,
                                  SpatiallySparseBatchInterface &output) {
  output.sub->features.resize(output.nSpatialSites *
                              output.featuresPresent.size());
  propForwardToMatrixMultiply(input.sub->features.dPtr(),
                              output.sub->features.dPtr(), output.rules.dPtr(),
                              output.nSpatialSites * fs,
                              input.featuresPresent.size(), memStream);
}
void ConvolutionalLayer::backwards(SpatiallySparseBatch &batch,
                                   SpatiallySparseBatchInterface &input,
                                   SpatiallySparseBatchInterface &output,
                                   float learningRate, float momentum) {
  if (input.backpropErrors) {
    input.sub->dfeatures.resize(input.nSpatialSites *
                                input.featuresPresent.size());
    input.sub->dfeatures.setZero(memStream);
    propBackwardFromMatrixMultiply(
        input.sub->dfeatures.dPtr(), output.sub->dfeatures.dPtr(),
        output.rules.dPtr(), output.nSpatialSites * fs,
        input.featuresPresent.size(), memStream);
  }
}
int ConvolutionalLayer::calculateInputSpatialSize(int outputSpatialSize) {
  outSpatialSize = outputSpatialSize;
  inSpatialSize = filterSize + (outputSpatialSize - 1) * filterStride;
  std::cout << "-(C" << filterSize;
  if (filterStride != 1)
    std::cout << "/" << filterStride;
  std::cout << ")-" << inSpatialSize;
  return inSpatialSize;
}
