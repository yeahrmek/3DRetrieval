#include "hip/hip_runtime.h"
// Performs a real convolution, used for the input layer. Other convolutions are
// implemented using ConvolutionLayer+NetworkInNetworkLayer

// 1) can use bz in dMultiply_Input_Weights_Output to access more
// output.nSpatialSites (4096*4096*32)?
// by+ty<outputNSpatialSites -> 4096*bz+by+ty<outputNSpatialSites, etc

#include "NetworkInNetworkLayer.h"
#include "ReallyConvolutionalLayer.h"
#include <iostream>
#include <vector>
#include <cassert>
#include "utilities.h"
#include "SigmoidLayer.h"
#include "Regions.h"

// Assume fs <=2*KERNELBLOCKSIZE

// // dMultiply_Input_Weights_Output <<<
// //
// dim3(output.featuresPresent.size()/KERNELBLOCKSIZE,(output.nSpatialSites+KERNELBLOCKSIZE-1)/KERNELBLOCKSIZE),
// //  dim3(KERNELBLOCKSIZE,KERNELBLOCKSIZE),0,memStream.stream>>>
// //
// (input.sub->features.dPtr(),w.dPtr(),b.dPtr(),output.rules.dPtr(),output.sub->features.dPtr(),
// //    input.featuresPresent.size(),output.featuresPresent.size(), fs,
// output.nSpatialSites,leaky);
__global__ void dMultiply_Input_Weights_Output(float *inFeatures, float *W,
                                               float *B, int *rules,
                                               float *outFeatures, int nIn,
                                               int nOut, int fs,
                                               int outputNSpatialSites,
                                               float leaky, float shrink = 1) {
  __shared__ float As[KERNELBLOCKSIZE][KERNELBLOCKSIZE];
  __shared__ float Bs[KERNELBLOCKSIZE][KERNELBLOCKSIZE];
  __shared__ int
      r[KERNELBLOCKSIZE][2 * KERNELBLOCKSIZE]; // Assume fs <=2*KERNELBLOCKSIZE
  int bx = blockIdx.x * KERNELBLOCKSIZE;
  int by = blockIdx.y * KERNELBLOCKSIZE;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  float acc = B[bx + tx];
  r[ty][tx] = (tx < fs and by + ty < outputNSpatialSites)
                  ? rules[(by + ty) * fs + tx]
                  : -1;
  r[ty][tx + KERNELBLOCKSIZE] =
      (tx + KERNELBLOCKSIZE < fs and by + ty < outputNSpatialSites)
          ? rules[(by + ty) * fs + tx + KERNELBLOCKSIZE]
          : -1;
  __syncthreads();
  for (int k = 0; k < nIn * fs; k += KERNELBLOCKSIZE) {
    int n = min(KERNELBLOCKSIZE, nIn * fs - k);
    int f = (k + tx) / nIn;
    int ff = (k + tx) % nIn;
    // int r=(tx<n and by+ty<outputNSpatialSites)?rules[(by+ty)*fs+f]:-1; /////
    // As[ty][tx]=(r>=0)?inFeatures[r*nIn+(ff)]:0; ////////
    As[ty][tx] = (r[ty][f] >= 0) ? inFeatures[r[ty][f] * nIn + (ff)] : 0;
    Bs[ty][tx] = (ty < n) ? W[(k + ty) * nOut + (bx + tx)] : 0;
    __syncthreads();
    for (int l = 0; l < n; l++)
      acc += As[ty][l] * Bs[l][tx];
    __syncthreads();
  }
  acc *= shrink;
  if (by + ty < outputNSpatialSites)
    outFeatures[(by + ty) * nOut + (bx + tx)] = (acc > 0) ? acc : (acc * leaky);
}

/************************************************************************************/
// // dMultiply_dOutput_WT_dInput
// //   <<<
// //
// dim3((input.featuresPresent.size()*fs+KERNELBLOCKSIZE-1)/KERNELBLOCKSIZE,(output.nSpatialSites+KERNELBLOCKSIZE-1)/KERNELBLOCKSIZE),
// //   dim3(KERNELBLOCKSIZE,KERNELBLOCKSIZE)
// //   ,0,memStream.stream>>>
__global__ void dMultiply_dOutput_WT_dInput(float *dOutFeatures, float *W,
                                            float *dInFeatures, int *rules,
                                            int nIn, int nOut, int fs,
                                            int outputNSpatialSites) {
  __shared__ float As[KERNELBLOCKSIZE][KERNELBLOCKSIZE];
  __shared__ float Bs[KERNELBLOCKSIZE][KERNELBLOCKSIZE];
  int bx = blockIdx.x * KERNELBLOCKSIZE;
  int by = blockIdx.y * KERNELBLOCKSIZE;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  float acc = 0;
  int n = min(KERNELBLOCKSIZE, nIn * fs - bx);
  for (int k = 0; k < nOut; k += KERNELBLOCKSIZE) {
    As[ty][tx] = (by + ty < outputNSpatialSites)
                     ? dOutFeatures[(by + ty) * nOut + k + tx]
                     : 0;
    Bs[tx][ty] = (ty < n) ? W[(bx + ty) * nOut + k + tx] : 0;
    __syncthreads();
    for (int l = 0; l < n; l++)
      acc += As[ty][l] * Bs[l][tx];
    __syncthreads();
  }
  int f = (bx + tx) / nIn;
  int ff = (bx + tx) % nIn;
  if (by + ty < outputNSpatialSites && f + tx < nIn * fs) {
    int r = rules[(by + ty) * fs + f];
    // dInFeatures[r*nIn+ff]+=acc;
    atomicAdd(&dInFeatures[r * nIn + ff], acc);
  }
}
/************************************************************************************/
// // dMultiply_InputT_dOutput_dWeights
// //   <<<
// //   dim3(output.featuresPresent.size()/KERNELBLOCKSIZE,
// //    (input.featuresPresent.size()*fs+KERNELBLOCKSIZE-1)/KERNELBLOCKSIZE,
// //    (output.nSpatialSites+KERNELBLOCKSIZE-1)/KERNELBLOCKSIZE),
// //   dim3(KERNELBLOCKSIZE,KERNELBLOCKSIZE)
// //   ,0,memStream.stream>>>
__global__ void dMultiply_InputT_dOutput_dWeights(float *inFeatures, int *rules,
                                                  float *dOutFeatures,
                                                  float *dW, int nIn, int nOut,
                                                  int fs,
                                                  int outputNSpatialSites) {
  __shared__ float As[KERNELBLOCKSIZE][KERNELBLOCKSIZE];
  __shared__ float Bs[KERNELBLOCKSIZE][KERNELBLOCKSIZE];
  int bx = blockIdx.x * KERNELBLOCKSIZE;
  int by = blockIdx.y * KERNELBLOCKSIZE;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  float acc = 0;
  int f = (by + ty) / nIn;
  int ff = (by + ty) % nIn;
  int k = blockIdx.z * KERNELBLOCKSIZE;
  {
    int n = min(KERNELBLOCKSIZE, outputNSpatialSites - k);
    int r = (tx < n and f < fs) ? rules[(k + tx) * fs + f] : -1;
    As[ty][tx] = (r >= 0) ? inFeatures[r * nIn + ff] : 0;
    Bs[ty][tx] = (ty < n) ? dOutFeatures[(k + ty) * nOut + bx + tx] : 0;
    __syncthreads();
    for (int l = 0; l < n; l++)
      acc += As[ty][l] * Bs[l][tx];
    __syncthreads();
  }
  if (f < fs)
    atomicAdd(&dW[(by + ty) * nOut + bx + tx], acc);
}

/************************************************************************************/
// Matrix is (nIn*fs)x(nOut)
// Shrink to (nInDropout*fs)*(nOut)
// inFeaturesPresent has length nInDropout
// outFeaturesPresent has length nOutDropout
__global__ void dShrinkMatrixForDropout(float *m, float *md,
                                        int *inFeaturesPresent,
                                        int *outFeaturesPresent, int nOut,
                                        int nOutDropout, int nIn,
                                        int nInDropout) {
  int i = blockIdx.x * nOutDropout;
  int ii = inFeaturesPresent[blockIdx.x % nInDropout] * nOut +
           (blockIdx.x / nInDropout) * nIn * nOut;
  for (int j = threadIdx.x; j < nOutDropout; j += KERNELBLOCKSIZE) {
    int jj = outFeaturesPresent[j];
    md[i + j] = m[ii + jj];
  }
}
__global__ void dGradientDescentShrunkMatrix(
    float *d_delta, float *d_momentum, float *d_weights, int nOut,
    int nOutDropout, int nIn, int nInDropout, int *inFeaturesPresent,
    int *outFeaturesPresent, float learningRate, float momentum) {
  int i = blockIdx.x * nOutDropout;
  int ii = inFeaturesPresent[blockIdx.x % nInDropout] * nOut +
           (blockIdx.x / nInDropout) * nIn * nOut;
  for (int j = threadIdx.x; j < nOutDropout; j += KERNELBLOCKSIZE) {
    int jj = outFeaturesPresent[j];
    // NAG light
    d_weights[ii + jj] -= d_momentum[ii + jj] * momentum;
    d_momentum[ii + jj] = momentum * d_momentum[ii + jj] -
                          learningRate * (1 - momentum) * d_delta[i + j];
    d_weights[ii + jj] =
        d_weights[ii + jj] + d_momentum[ii + jj] * (1 + momentum);
  }
}

ReallyConvolutionalLayer::ReallyConvolutionalLayer(
    cudaMemStream &memStream, int nFeaturesIn, int nFeaturesOut, int filterSize,
    int filterStride, int dimension, ActivationFunction fn, float dropout,
    int minActiveInputs, float poolingToFollow)
    : SpatiallySparseLayer(memStream), nFeaturesIn(nFeaturesIn),
      nFeaturesOut(nFeaturesOut), filterSize(filterSize),
      filterStride(filterStride), dimension(dimension), fn(fn),
      dropout(dropout), fs(ipow(filterSize, dimension)),
      W(true, nFeaturesIn * fs * nFeaturesOut),
      MW(true, nFeaturesIn * fs * nFeaturesOut), B(true, nFeaturesOut),
      MB(true, nFeaturesOut), minActiveInputs(minActiveInputs) {
  std::cout << "Convolution " << filterSize << "^" << dimension << "x"
            << nFeaturesIn << "=" << fs *nFeaturesIn << "->" << nFeaturesOut;
  if (filterStride > 1)
    std::cout << " stride " << filterStride;
  std::cout << " dropout=" << dropout << " " << sigmoidNames[fn] << std::endl;
  float scale = pow(
      6.0f / (nFeaturesIn * fs +
              nFeaturesOut *
                  powf(filterSize * 1.0 / filterStride / poolingToFollow, 2)),
      0.5f);
  W.setUniform(-scale, scale);
  MW.setZero();
  B.setZero();
  MB.setZero();
  switch (fn) {
  case RELU:
    leaky = 0;
    break;
  case LEAKYRELU:
    leaky = 0.01;
    break;
  case VLEAKYRELU:
    leaky = 0.333;
    break;
  default:
    assert(0);
  }
}
void ReallyConvolutionalLayer::preprocess(
    SpatiallySparseBatch &batch, SpatiallySparseBatchInterface &input,
    SpatiallySparseBatchInterface &output) {
  output.nFeatures = nFeaturesOut;
  assert(input.nFeatures == nFeaturesIn);
  assert(input.spatialSize >= filterSize);
  assert((input.spatialSize - filterSize) % filterStride == 0);
  output.spatialSize = (input.spatialSize - filterSize) / filterStride + 1;
  output.nSpatialSites = 0;
  output.grids.resize(batch.batchSize);
  output.backpropErrors = true;
  RegularSquareRegions regions(inSpatialSize, outSpatialSize, dimension,
                               filterSize, filterStride);
  for (int item = 0; item < batch.batchSize; item++) {
    gridRules(input.grids[item], output.grids[item], regions,
              output.nSpatialSites, output.rules.hVector(), minActiveInputs);
  }
  int o = nFeaturesOut * (batch.type == TRAINBATCH ? (1.0f - dropout) : 1.0f);
  output.featuresPresent.hVector() = rng.NchooseM(nFeaturesOut, o);
}
void ReallyConvolutionalLayer::forwards(SpatiallySparseBatch &batch,
                                        SpatiallySparseBatchInterface &input,
                                        SpatiallySparseBatchInterface &output) {
  output.sub->features.resize(output.nSpatialSites *
                              output.featuresPresent.size());
  if (batch.type == TRAINBATCH and
      nFeaturesIn + nFeaturesOut >
          input.featuresPresent.size() + output.featuresPresent.size()) {
    w.resize(input.featuresPresent.size() * fs * output.featuresPresent.size());
    dShrinkMatrixForDropout << <input.featuresPresent.size() * fs,
                                KERNELBLOCKSIZE, 0, memStream.stream>>>
        (W.dPtr(), w.dPtr(), input.featuresPresent.dPtr(),
         output.featuresPresent.dPtr(), output.nFeatures,
         output.featuresPresent.size(), input.nFeatures,
         input.featuresPresent.size());
    cudaCheckError();
    b.resize(output.featuresPresent.size());
    dShrinkVectorForDropout << <1, NTHREADS, 0, memStream.stream>>>
        (B.dPtr(), b.dPtr(), output.featuresPresent.dPtr(), output.nFeatures,
         output.featuresPresent.size());
    cudaCheckError();
    dMultiply_Input_Weights_Output
            << <dim3(output.featuresPresent.size() / KERNELBLOCKSIZE,
                     (output.nSpatialSites + KERNELBLOCKSIZE - 1) /
                         KERNELBLOCKSIZE),
                dim3(KERNELBLOCKSIZE, KERNELBLOCKSIZE), 0, memStream.stream>>>
        (input.sub->features.dPtr(), w.dPtr(), b.dPtr(), output.rules.dPtr(),
         output.sub->features.dPtr(), input.featuresPresent.size(),
         output.featuresPresent.size(), fs, output.nSpatialSites, leaky);
    cudaCheckError();
  } else {
    dMultiply_Input_Weights_Output
            << <dim3(output.featuresPresent.size() / KERNELBLOCKSIZE,
                     (output.nSpatialSites + KERNELBLOCKSIZE - 1) /
                         KERNELBLOCKSIZE),
                dim3(KERNELBLOCKSIZE, KERNELBLOCKSIZE), 0, memStream.stream>>>
        (input.sub->features.dPtr(), W.dPtr(), B.dPtr(), output.rules.dPtr(),
         output.sub->features.dPtr(), input.featuresPresent.size(),
         output.featuresPresent.size(), fs, output.nSpatialSites, leaky,
         1.0f - dropout);
    cudaCheckError();
  }
  multiplyAddCount += (__int128_t)output.nSpatialSites *
                      input.featuresPresent.size() * fs *
                      output.featuresPresent.size();
  cudaCheckError();
}
void ReallyConvolutionalLayer::scaleWeights(
    SpatiallySparseBatchInterface &input, SpatiallySparseBatchInterface &output,
    float &scalingUnderneath, bool topLayer) {
  assert(input.sub->features.size() > 0);
  assert(output.sub->features.size() > 0); // call after forwards(...)
  float scale = output.sub->features.meanAbs();
  std::cout << "featureScale:" << scale << std::endl;
  if (topLayer) {
    scale = 1;
  } else {
    scale = powf(
        scale,
        -0.1); // 0.7978846 = sqrt(2/pi) = mean of the half normal distribution
  }
  W.multiplicativeRescale(scale / scalingUnderneath);
  B.multiplicativeRescale(scale);
  MW.multiplicativeRescale(scale / scalingUnderneath);
  MB.multiplicativeRescale(scale);
  scalingUnderneath = scale;
}

void ReallyConvolutionalLayer::backwards(SpatiallySparseBatch &batch,
                                         SpatiallySparseBatchInterface &input,
                                         SpatiallySparseBatchInterface &output,
                                         float learningRate, float momentum) {
  applySigmoidBackProp(output, output, fn, memStream);
  dw.resize(input.featuresPresent.size() * fs * output.featuresPresent.size());
  dw.setZero(
      memStream); //////////////////////////////////////////////////////////////////
  db.resize(output.featuresPresent.size());
  db.setZero(memStream);
  columnSum(output.sub->dfeatures.dPtr(), db.dPtr(), output.nSpatialSites,
            output.featuresPresent.size(), memStream);
  cudaCheckError();
  dMultiply_InputT_dOutput_dWeights
          << <dim3(output.featuresPresent.size() / KERNELBLOCKSIZE,
                   (input.featuresPresent.size() * fs + KERNELBLOCKSIZE - 1) /
                       KERNELBLOCKSIZE,
                   (output.nSpatialSites + KERNELBLOCKSIZE - 1) /
                       KERNELBLOCKSIZE),
              dim3(KERNELBLOCKSIZE, KERNELBLOCKSIZE), 0, memStream.stream>>>
      (input.sub->features.dPtr(), output.rules.dPtr(),
       output.sub->dfeatures.dPtr(), dw.dPtr(), input.featuresPresent.size(),
       output.featuresPresent.size(), fs, output.nSpatialSites);
  multiplyAddCount += (__int128_t)output.nSpatialSites *
                      input.featuresPresent.size() * fs *
                      output.featuresPresent.size();
  cudaCheckError();

  if (nFeaturesIn + nFeaturesOut >
      input.featuresPresent.size() + output.featuresPresent.size()) {
    if (input.backpropErrors) {
      input.sub->dfeatures.resize(input.nSpatialSites *
                                  input.featuresPresent.size());
      input.sub->dfeatures.setZero(memStream);
      dMultiply_dOutput_WT_dInput << <
          dim3((input.featuresPresent.size() * fs + KERNELBLOCKSIZE - 1) /
                   KERNELBLOCKSIZE,
               (output.nSpatialSites + KERNELBLOCKSIZE - 1) / KERNELBLOCKSIZE),
          dim3(KERNELBLOCKSIZE, KERNELBLOCKSIZE), 0, memStream.stream>>>
          (output.sub->dfeatures.dPtr(), w.dPtr(), input.sub->dfeatures.dPtr(),
           output.rules.dPtr(), input.featuresPresent.size(),
           output.featuresPresent.size(), fs, output.nSpatialSites);
      multiplyAddCount += (__int128_t)output.nSpatialSites *
                          input.featuresPresent.size() * fs *
                          output.featuresPresent.size();
      cudaCheckError();
    }
    dGradientDescentShrunkMatrix << <input.featuresPresent.size(),
                                     KERNELBLOCKSIZE, 0, memStream.stream>>>
        (dw.dPtr(), MW.dPtr(), W.dPtr(), output.nFeatures,
         output.featuresPresent.size(), input.featuresPresent.dPtr(),
         output.featuresPresent.dPtr(), learningRate, momentum);

    dGradientDescentShrunkVector << <1, NTHREADS, 0, memStream.stream>>>
        (db.dPtr(), MB.dPtr(), B.dPtr(), output.nFeatures,
         output.featuresPresent.size(), output.featuresPresent.dPtr(),
         learningRate, momentum);
  } else {
    if (input.backpropErrors) {
      input.sub->dfeatures.resize(input.nSpatialSites *
                                  input.featuresPresent.size());
      input.sub->dfeatures.setZero(memStream);
      dMultiply_dOutput_WT_dInput << <
          dim3((input.featuresPresent.size() * fs + KERNELBLOCKSIZE - 1) /
                   KERNELBLOCKSIZE,
               (output.nSpatialSites + KERNELBLOCKSIZE - 1) / KERNELBLOCKSIZE),
          dim3(KERNELBLOCKSIZE, KERNELBLOCKSIZE), 0, memStream.stream>>>
          (output.sub->dfeatures.dPtr(), W.dPtr(), input.sub->dfeatures.dPtr(),
           output.rules.dPtr(), input.featuresPresent.size(),
           output.featuresPresent.size(), fs, output.nSpatialSites);
      multiplyAddCount += (__int128_t)output.nSpatialSites *
                          input.featuresPresent.size() * fs *
                          output.featuresPresent.size();
      cudaCheckError();
    }
    dGradientDescent << <nFeaturesIn, KERNELBLOCKSIZE, 0, memStream.stream>>>
        (dw.dPtr(), MW.dPtr(), W.dPtr(), nFeaturesOut, learningRate, momentum);
    dGradientDescent << <1, KERNELBLOCKSIZE, 0, memStream.stream>>>
        (db.dPtr(), MB.dPtr(), B.dPtr(), nFeaturesOut, learningRate, momentum);
  }
  cudaCheckError();
}
int ReallyConvolutionalLayer::calculateInputSpatialSize(int outputSpatialSize) {
  outSpatialSize = outputSpatialSize;
  inSpatialSize = filterSize + (outputSpatialSize - 1) * filterStride;
  std::cout << "-(C" << filterSize;
  if (filterStride != 1)
    std::cout << "/" << filterStride;
  std::cout << ")-" << inSpatialSize;
  return inSpatialSize;
}
