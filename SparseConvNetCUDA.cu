#include "hip/hip_runtime.h"
#include "SparseConvNetCUDA.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <cassert>
#include <algorithm>
#include "utilities.h"
#include "SigmoidLayer.h"
#include "NetworkInNetworkLayer.h"
#include "NetworkInNetworkPReLULayer.h"
#include "ConvolutionalLayer.h"
#include "ReallyConvolutionalLayer.h"
#include "ConvolutionalTriangularLayer.h"
#include "MaxPoolingLayer.h"
#include "MaxPoolingTriangularLayer.h"
#include "TerminalPoolingLayer.h"
#include "IndexLearnerLayer.h"
#include "SoftmaxClassifier.h"
#include "BatchProducer.h"
#include "SpatiallySparseDataset.h"

SparseConvNetCUDA::SparseConvNetCUDA(int dimension, int nInputFeatures,
                                     int nClasses, int pciBusID, int nTop,
                                     int nBatchProducerThreads)
    : deviceID(initializeGPU(pciBusID)), dimension(dimension),
      nInputFeatures(nInputFeatures), nClasses(nClasses), nTop(nTop),
      nBatchProducerThreads(nBatchProducerThreads) {
  assert(nBatchProducerThreads <= N_MAX_BATCH_PRODUCER_THREADS);
  std::cout << "Sparse CNN - dimension=" << dimension
            << " nInputFeatures=" << nInputFeatures << " nClasses=" << nClasses
            << std::endl;
  nOutputFeatures = nInputFeatures;
  // Set up a pool of SpatiallySparseBatches
  for (int c = 0; c < nBatchProducerThreads; c++) {
    initialSubInterfaces.push_back(new SpatiallySparseBatchSubInterface());
    batchPool.emplace_back(initialSubInterfaces.back());
  }
  cublasError(hipblasCreate(&cublasHandle), __FILE__, __LINE__);
  cublasError(hipblasSetStream(cublasHandle, memStream.stream));
}
SparseConvNetCUDA::~SparseConvNetCUDA() {
  for (auto p : initialSubInterfaces)
    delete p;
  for (auto p : sharedSubInterfaces)
    delete p;
  cublasError(hipblasDestroy(cublasHandle), __FILE__, __LINE__);
}
void SparseConvNetCUDA::addLearntLayer(int nFeatures,
                                       ActivationFunction activationFn,
                                       float dropout, float alpha) {
  if (activationFn != SOFTMAX)
    nFeatures = std::max(KERNELBLOCKSIZE, intRound(nFeatures, KERNELBLOCKSIZE));
  if (dropout > 0)
    dropout = 1 -
              (intRound(nFeatures * (1 - dropout), KERNELBLOCKSIZE) + 0.01f) *
                  1.0f / nFeatures;
  if (dropout < 0)
    dropout = 0;
  std::cout << layers.size() << ":";
  if (activationFn == PRELU)
    layers.push_back(new NetworkInNetworkPReLULayer(
        memStream, cublasHandle, nOutputFeatures, nFeatures, dropout, alpha));
  else
    layers.push_back(new NetworkInNetworkLayer(memStream, cublasHandle,
                                               nOutputFeatures, nFeatures,
                                               dropout, activationFn, alpha));
  nOutputFeatures = nFeatures;
}
void SparseConvNetCUDA::addNetworkInNetworkLayer(
    int nFeatures, ActivationFunction activationFn, float dropout) {
  addLearntLayer(nFeatures, activationFn, dropout, 1.0f);
}
void SparseConvNetCUDA::addConvolutionalLayer(int nFeatures, int filterSize,
                                              int filterStride,
                                              ActivationFunction activationFn,
                                              float dropout,
                                              int minActiveInputs,
                                              float poolingToFollow) {
  if (false and layers.size() == 0) { // Use for 0-th layer??
    std::cout << layers.size() << ":";
    layers.push_back(new ReallyConvolutionalLayer(
        memStream, nOutputFeatures, nFeatures, filterSize, filterStride,
        dimension, activationFn, dropout, minActiveInputs, poolingToFollow));
    nOutputFeatures = nFeatures;
  } else {
    if (filterSize > 1) {
      std::cout << layers.size() << ":";
      layers.push_back(
          new ConvolutionalLayer(memStream, filterSize, filterStride, dimension,
                                 nOutputFeatures, minActiveInputs));
      nOutputFeatures *= ipow(filterSize, dimension);
    }
    addLearntLayer(nFeatures, activationFn, dropout,
                   powf(filterSize * 1.0 / filterStride / poolingToFollow, 2));
  }
}
void SparseConvNetCUDA::addLeNetLayerMP(int nFeatures, int filterSize,
                                        int filterStride, int poolSize,
                                        int poolStride,
                                        ActivationFunction activationFn,
                                        float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, poolSize);
  if (poolSize > 1) {
    std::cout << layers.size() << ":";
    layers.push_back(
        new MaxPoolingLayer(memStream, poolSize, poolStride, dimension));
  }
}
void SparseConvNetCUDA::addLeNetLayerROFMP(int nFeatures, int filterSize,
                                           int filterStride, int poolSize,
                                           float fmpShrink,
                                           ActivationFunction activationFn,
                                           float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, fmpShrink);
  if (fmpShrink > 1) {
    std::cout << layers.size() << ":";
    layers.push_back(new RandomOverlappingFractionalMaxPoolingLayer(
        memStream, poolSize, fmpShrink, dimension));
  }
}
void SparseConvNetCUDA::addLeNetLayerPOFMP(int nFeatures, int filterSize,
                                           int filterStride, int poolSize,
                                           float fmpShrink,
                                           ActivationFunction activationFn,
                                           float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, fmpShrink);
  if (fmpShrink > 1) {
    std::cout << layers.size() << ":";
    layers.push_back(new PseudorandomOverlappingFractionalMaxPoolingLayer(
        memStream, poolSize, fmpShrink, dimension));
  }
}
void SparseConvNetCUDA::addLeNetLayerRDFMP(int nFeatures, int filterSize,
                                           int filterStride, int poolSize,
                                           float fmpShrink,
                                           ActivationFunction activationFn,
                                           float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, fmpShrink);
  if (fmpShrink > 1) {
    std::cout << layers.size() << ":";
    layers.push_back(new RandomNonOverlappingFractionalMaxPoolingLayer(
        memStream, poolSize, fmpShrink, dimension));
  }
}
void SparseConvNetCUDA::addLeNetLayerPDFMP(int nFeatures, int filterSize,
                                           int filterStride, int poolSize,
                                           float fmpShrink,
                                           ActivationFunction activationFn,
                                           float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, fmpShrink);
  if (fmpShrink > 1) {
    std::cout << layers.size() << ":";
    layers.push_back(new PseudorandomNonOverlappingFractionalMaxPoolingLayer(
        memStream, poolSize, fmpShrink, dimension));
  }
}

void SparseConvNetCUDA::addTriangularConvolutionalLayer(
    int nFeatures, int filterSize, int filterStride,
    ActivationFunction activationFn, float dropout, int minActiveInputs,
    float poolingToFollow) {
  if (filterSize > 1) {
    std::cout << layers.size() << ":";
    layers.push_back(new ConvolutionalTriangularLayer(
        memStream, filterSize, filterStride, dimension, nOutputFeatures,
        minActiveInputs));
    nOutputFeatures *= triangleSize(filterSize, dimension);
  }
  addLearntLayer(nFeatures, activationFn, dropout,
                 powf(filterSize * 1.0 / filterStride / poolingToFollow, 2));
}
void SparseConvNetCUDA::addTriangularLeNetLayerMP(
    int nFeatures, int filterSize, int filterStride, int poolSize,
    int poolStride, ActivationFunction activationFn, float dropout,
    int minActiveInputs) {
  addTriangularConvolutionalLayer(nFeatures, filterSize, filterStride,
                                  activationFn, dropout, poolSize,
                                  minActiveInputs);
  if (poolSize > 1) {
    std::cout << layers.size() << ":";
    layers.push_back(new MaxPoolingTriangularLayer(memStream, poolSize,
                                                   poolStride, dimension));
  }
}

void SparseConvNetCUDA::addTerminalPoolingLayer(int poolSize, int S) {
  std::cout << layers.size() << ":";
  layers.push_back(new TerminalPoolingLayer(memStream, poolSize, S));
}

void SparseConvNetCUDA::addSoftmaxLayer() {
  addLearntLayer(nClasses, SOFTMAX, 0.0f, 1);
  inputSpatialSize = 1;
  std::cout << "Spatially sparse CNN with layer sizes: " << inputSpatialSize;
  for (int i = layers.size() - 1; i >= 0; i--) {
    inputSpatialSize = layers[i]->calculateInputSpatialSize(inputSpatialSize);
  }
  std::cout << std::endl;
  std::cout << "Input-field dimensions = " << inputSpatialSize;
  for (int i = 1; i < dimension; ++i)
    std::cout << "x" << inputSpatialSize;
  std::cout << std::endl;
}
void SparseConvNetCUDA::addIndexLearnerLayer() {
  std::cout << layers.size() << ":";
  layers.push_back(new IndexLearnerLayer(memStream, cublasHandle,
                                         nOutputFeatures, nClasses));
  std::cout << "Index Learner " << nOutputFeatures << "-> " << nClasses
            << std::endl;
  nOutputFeatures = nClasses; // "nClasses"=trainingSet.pictures.size()
  inputSpatialSize = 1;
  for (int i = layers.size() - 1; i >= 0; i--) {
    inputSpatialSize = layers[i]->calculateInputSpatialSize(inputSpatialSize);
  }
  std::cout << "Spatially sparse CNN: input size " << inputSpatialSize;
  for (int i = 1; i < dimension; ++i)
    std::cout << "x" << inputSpatialSize;
  std::cout << std::endl;
}
void SparseConvNetCUDA::processBatch(SpatiallySparseBatch &batch,
                                     float learningRate, float momentum,
                                     std::ofstream &f, std::ofstream &g) {
  if (batch.type == RESCALEBATCH) {
    float scalingUnderneath = 1;
    for (int i = 0; i < layers.size(); i++) {
      batch.interfaces[i + 1].sub->reset();
      layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
      std::cout << i << ":"
                << batch.interfaces[i].sub->features.size() * sizeof(float) /
                       (1 << 20) << "MB ";
      layers[i]->scaleWeights(batch.interfaces[i], batch.interfaces[i + 1],
                              scalingUnderneath, i == layers.size() - 1);
    }
  } else {
    for (int i = 0; i < layers.size(); i++) {
      batch.interfaces[i + 1].sub->reset();
      layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
    }
  }
  SoftmaxClassifier(batch.interfaces.back(), batch, nTop, memStream);
  if (batch.type == TRAINBATCH)
    for (int i = layers.size() - 1; i >= 0; i--) {
      layers[i]->backwards(batch, batch.interfaces[i], batch.interfaces[i + 1],
                           learningRate, momentum);
    }
  if (f)
    for (int j = 0; j < batch.predictions.size(); j++) {
      for (int k = 0; k < batch.predictions[j].size(); k++) {
        if (k > 0)
          f << " ";
        f << batch.predictions[j][k];
      }
      f << std::endl;
    }
  if (g)
    for (int j = 0; j < batch.predictions.size(); j++) {
      for (int k = 0; k < batch.probabilities[j].size(); k++) {
        if (k > 0)
          g << " ";
        g << batch.probabilities[j][k];
      }
      g << std::endl;
    }
}
float SparseConvNetCUDA::processDataset(SpatiallySparseDataset &dataset,
                                        int batchSize, float learningRate,
                                        float momentum) {
  assert(dataset.pictures.size() > 0);
  float errorRate = 0, nll = 0;
  multiplyAddCount = 0;
  auto start = std::chrono::system_clock::now();
  std::ofstream f, g;
  BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
  if (dataset.type == UNLABELEDBATCH) {
    f.open("unlabelledData.predictions");
    g.open("unlabelledData.probabilities");
  }
  while (SpatiallySparseBatch *batch = bp.nextBatch()) {
    processBatch(*batch, learningRate, momentum, f, g);
    errorRate += batch->mistakes * 1.0 / dataset.pictures.size();
    nll += batch->negativeLogLikelihood * 1.0 / dataset.pictures.size();
  }
  auto end = std::chrono::system_clock::now();
  auto diff =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  // for (int c = 0; c < nBatchProducerThreads; c++) {
  //   std::cout << "Batch " << c << "\n";
  //   std::cout << "labels " << batchPool[c].labels.size() << "\n";
  //   for (int i = 0; i < batchPool[c].interfaces.size(); ++i) {
  //     std::cout << i << ":" << (batchPool[c].interfaces[i].rules.size() >>
  //     18)
  //               << "\n";
  //     std::cout << i << ":"
  //               << (batchPool[c].interfaces[i].sub->features.size() >> 18)
  //               << "\n";
  //     std::cout << i << ":"
  //               << (batchPool[c].interfaces[i].sub->dfeatures.size() >> 18)
  //               << "\n";
  //     std::cout << i << ":"
  //               << (batchPool[c].interfaces[i].sub->poolingChoices.size() >>
  //               18)
  //               << "\n";
  //   }
  // }
  if (dataset.type != RESCALEBATCH)
    std::cout << dataset.name << " Mistakes:" << 100.0 * errorRate
              << "% NLL:" << nll << " MegaMultiplyAdds/sample:"
              << roundf(multiplyAddCount / dataset.pictures.size() / 1000000)
              << " time:" << diff / 1000000000L
              << "s GigaMultiplyAdds/s:" << roundf(multiplyAddCount / diff)
              << " rate:"
              << roundf(dataset.pictures.size() * 1000000000.0f / diff) << "/s"
              << std::endl;
  return nll;
}
void SparseConvNetCUDA::processDatasetRepeatTest(
    SpatiallySparseDataset &dataset, int batchSize, int nReps,
    std::string predictionsFilename, std::string confusionMatrixFilename) {
  assert(dataset.pictures.size() > 0);
  multiplyAddCount = 0;
  auto start = std::chrono::system_clock::now();
  std::vector<std::vector<int>> votes(dataset.pictures.size());
  std::vector<std::vector<float>> probs(dataset.pictures.size());
  for (int i = 0; i < dataset.pictures.size(); ++i) {
    votes[i].resize(dataset.nClasses);
    probs[i].resize(dataset.nClasses);
  }
  for (int rep = 1; rep <= nReps; ++rep) {
    BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
    while (SpatiallySparseBatch *batch = bp.nextBatch()) {
      std::ofstream f, g;
      processBatch(*batch, 0, 0, f, g);
      for (int i = 0; i < batch->batchSize; ++i) {
        int ii = batch->sampleNumbers[i];
        votes[ii][batch->predictions[i][0]]++;
        for (int j = 0; j < dataset.nClasses; ++j)
          probs[ii][j] += batch->probabilities[i][j];
      }
    }
    int errors = dataset.pictures.size();
    float nll = 0;
    for (int i = 0; i < dataset.pictures.size(); ++i) {
      std::vector<int> predictions = vectorTopIndices(probs[i], nTop);
      for (int j = 0; j < nTop; j++)
        if (predictions[j] == dataset.pictures[i]->label)
          errors--;
      nll -= log(
          std::max(probs[i][dataset.pictures[i]->label] / rep, (float)1.0e-15));
    }

    if (!predictionsFilename.empty()) {
      std::cout << predictionsFilename << std::endl;
      std::ofstream f(predictionsFilename.c_str());
      if (!dataset.header.empty())
        f << dataset.header << std::endl;
      for (int i = 0; i < dataset.pictures.size(); ++i) {
        f << dataset.pictures[i]->identify();
        if (dataset.type != UNLABELEDBATCH)
          f << "," << dataset.pictures[i]->label;
        for (int j = 0; j < dataset.nClasses; ++j)
          f << "," << probs[i][j] / rep;
        f << std::endl;
      }
    }
    if (!confusionMatrixFilename.empty()) {
      std::vector<float> cm(dataset.nClasses * dataset.nClasses);
      for (int i = 0; i < dataset.pictures.size(); ++i)
        for (int j = 0; j < dataset.nClasses; ++j)
          cm[dataset.pictures[i]->label * dataset.nClasses + j] +=
              probs[i][j] / rep;
      std::ofstream f(confusionMatrixFilename.c_str());
      for (int i = 0; i < dataset.nClasses; ++i) {
        for (int j = 0; j < dataset.nClasses; ++j) {
          f << cm[i * dataset.nClasses + j] << " ";
        }
        f << std::endl;
      }
    }
    auto end = std::chrono::system_clock::now();
    auto diff = std::chrono::duration_cast<std::chrono::nanoseconds>(
                    end - start).count();
    std::cout << dataset.name << " rep " << rep << "/" << nReps
              << " Mistakes: " << 100.0 * errors / dataset.pictures.size()
              << "% NLL " << nll / dataset.pictures.size()
              << " MegaMultiplyAdds/sample:"
              << roundf(multiplyAddCount / dataset.pictures.size() / 1000000)
              << " time:" << diff / 1000000000L
              << "s GigaMultiplyAdds/s:" << roundf(multiplyAddCount / diff)
              << " rate:"
              << roundf(dataset.pictures.size() * 1000000000.0f / diff) << "/s"
              << std::endl;
  }
}
void SparseConvNetCUDA::loadWeights(std::string baseName, int epoch,
                                    bool momentum, int firstNlayers) {
  std::string filename = std::string(baseName) + std::string("_epoch-") +
                         std::to_string(epoch) + std::string(".cnn");
  std::ifstream f;
  f.open(filename.c_str(), std::ios::out | std::ios::binary);
  if (f) {
    std::cout << "Loading network parameters from " << filename << std::endl;
  } else {
    std::cout << "Cannot find " << filename << std::endl;
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < std::min((int)layers.size(), firstNlayers); i++)
    layers[i]->loadWeightsFromStream(f, momentum);
  if (inputNormalizingConstants.size() > 0)
    f.read((char *)&inputNormalizingConstants[0],
           sizeof(float) * inputNormalizingConstants.size());
  f.close();
}
void SparseConvNetCUDA::saveWeights(std::string baseName, int epoch,
                                    bool momentum) {
  std::string filename = std::string(baseName) + std::string("_epoch-") +
                         std::to_string(epoch) + std::string(".cnn");
  std::ofstream f;
  f.open(filename.c_str(), std::ios::binary);
  if (f) {
    for (int i = 0; i < layers.size(); i++)
      layers[i]->putWeightsToStream(f, momentum);
    if (inputNormalizingConstants.size() > 0)
      f.write((char *)&inputNormalizingConstants[0],
              sizeof(float) * inputNormalizingConstants.size());
    f.close();
  } else {
    std::cout << "Cannot write " << filename << std::endl;
    exit(EXIT_FAILURE);
  }
}
void SparseConvNetCUDA::processIndexLearnerBatch(SpatiallySparseBatch &batch,
                                                 float learningRate,
                                                 float momentum,
                                                 std::ofstream &f) {
  int n = layers.size();
  for (int i = 0; i < n - 1; i++) // Stop 1 early (unless it is a training
                                  // batch)
    layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
  if (f.is_open()) {
    assert(batch.interfaces[n - 1].nFeatures ==
           batch.interfaces[n - 1].featuresPresent.size());
    for (int i = 0; i < batch.batchSize; i++) {
      f << batch.sampleNumbers[i] << " " << batch.labels.hVector()[i];
      for (int j = 0; j < batch.interfaces[n - 1].nFeatures; j++)
        f << " "
          << batch.interfaces[n - 1]
                 .sub->features
                 .hVector()[i * batch.interfaces[n - 1].nFeatures + j];
      f << std::endl;
    }
  }
  if (batch.type == TRAINBATCH) {
    static_cast<IndexLearnerLayer *>(layers[n - 1])->indexLearnerIndices =
        batch.sampleNumbers;
    layers[n - 1]->forwards(batch, batch.interfaces[n - 1],
                            batch.interfaces[n]);
    IndexLearner(batch.interfaces[n], batch, nTop, memStream);
    for (int i = n - 1; i >= 0; i--)
      layers[i]->backwards(batch, batch.interfaces[i], batch.interfaces[i + 1],
                           learningRate, momentum);
  }
}
float SparseConvNetCUDA::processIndexLearnerDataset(
    SpatiallySparseDataset &dataset, int batchSize, float learningRate,
    float momentum) {
  assert(dataset.pictures.size() > 0);
  float errorRate = 0, nll = 0;
  auto start = std::chrono::system_clock::now();
  multiplyAddCount = 0;
  std::ofstream f;
  BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
  if (dataset.type != TRAINBATCH) {
    std::string filename = dataset.name + ".features";
    f.open(filename.c_str());
  }
  while (SpatiallySparseBatch *batch = bp.nextBatch()) {
    processIndexLearnerBatch(*batch, learningRate, momentum, f);
    errorRate += batch->mistakes * 1.0 / dataset.pictures.size();
    nll += batch->negativeLogLikelihood * 1.0 / dataset.pictures.size();
  }
  auto end = std::chrono::system_clock::now();
  auto diff =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  if (dataset.type == TRAINBATCH)
    std::cout << dataset.name << " Mistakes:" << 100 * errorRate
              << "% NLL:" << nll << " MegaMultiplyAdds/sample:"
              << roundf(multiplyAddCount / dataset.pictures.size() / 1000000)
              << " time:" << diff / 1000000000L
              << "s GigaMultiplyAdds/s:" << roundf(multiplyAddCount / diff)
              << " rate:"
              << roundf(dataset.pictures.size() * 1000000000.0f / diff) << "/s"
              << std::endl;
  return nll;
}
void SparseConvNetCUDA::processBatchDumpTopLevelFeaturess(
    SpatiallySparseBatch &batch, std::ofstream &f) { // editted: test
  int n = layers.size();
  for (int i = 0; i < layers.size() - 1; i++) {
    layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
  }
  assert(batch.interfaces[n - 1].nFeatures ==
         batch.interfaces[n - 1].featuresPresent.size());
  for (int i = 0; i < batch.batchSize; i++) {
    f << batch.sampleNumbers[i] << " " << batch.labels.hVector()[i];
    for (int j = 0; j < batch.interfaces[n - 1].nFeatures; j++)
      f << " "
        << batch.interfaces[n - 1]
               .sub->features
               .hVector()[i * batch.interfaces[n - 1].nFeatures + j];
    f << std::endl;
  }
}
void SparseConvNetCUDA::processDatasetDumpTopLevelFeatures(
    SpatiallySparseDataset &dataset, int batchSize, int reps) {
  assert(dataset.pictures.size() > 0);
  std::ofstream f;
  assert(dataset.type != TRAINBATCH);
  std::string filename = dataset.name + ".features";
  f.open(filename.c_str());
  for (int i = 0; i < reps; i++) {
    BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
    while (SpatiallySparseBatch *batch = bp.nextBatch()) {
      processBatchDumpTopLevelFeaturess(*batch, f);
    }
  }
}

void SparseConvNetCUDA::calculateInputRegularizingConstants(
    SpatiallySparseDataset dataset) { // make copy of the dataset
  inputNormalizingConstants.resize(
      0); // Make sure input features rescaling is turned off.
  std::cout << "Using " << std::min(10000, (int)dataset.pictures.size())
            << " out of " << dataset.pictures.size()
            << " training samples to calculate regularizing constants."
            << std::endl;
  if (dataset.pictures.size() > 10000)
    dataset.pictures.resize(10000);
  dataset.type = TESTBATCH; // pretend it is a test batch to turn off dropout
                            // and training data augmentation
  BatchProducer bp(*this, dataset, inputSpatialSize, 100);
  std::vector<float> c(nInputFeatures, 0);
  while (SpatiallySparseBatch *batch = bp.nextBatch()) {
    batch->interfaces[0].sub->features.copyToCPUAsync(memStream);
    std::vector<float> &features = batch->interfaces[0].sub->features.hVector();
    for (int i = 0; i < features.size(); ++i)
      c[i % nInputFeatures] =
          std::max(c[i % nInputFeatures], std::fabs(features[i]));
  }
  for (int i = 0; i < nInputFeatures; ++i) {
    inputNormalizingConstants.push_back(c[i] > 0 ? 1.0f / c[i] : 0);
    std::cout << inputNormalizingConstants.back() << " ";
  }
  std::cout << std::endl;
}
